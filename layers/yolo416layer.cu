#include "hip/hip_runtime.h"
#include "yolo416layer.h"

using namespace Yolo416;

namespace nvinfer1
{
    Yolo416LayerPlugin::Yolo416LayerPlugin()
    {
        mClassCount = CLASS_NUM;
        mYoloKernel.clear();
        mYoloKernel.push_back(yolo1);
        mYoloKernel.push_back(yolo2);
        mYoloKernel.push_back(yolo3);

        mKernelCount = mYoloKernel.size();

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        for(int ii = 0; ii < mKernelCount; ii ++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii],AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    
    Yolo416LayerPlugin::~Yolo416LayerPlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    Yolo416LayerPlugin::Yolo416LayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(mYoloKernel.data(),d,kernelSize);
        d += kernelSize;

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        for(int ii = 0; ii < mKernelCount; ii ++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii],AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }

        assert(d == a + length);
    }

    void Yolo416LayerPlugin::serialize(void* buffer) const
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(d,mYoloKernel.data(),kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t Yolo416LayerPlugin::getSerializationSize() const
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount)  + sizeof(Yolo416::YoloKernel) * mYoloKernel.size();
    }

    int Yolo416LayerPlugin::initialize()
    { 
        return 0;
    }
    
    Dims Yolo416LayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalsize = MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void Yolo416LayerPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* Yolo416LayerPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType Yolo416LayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool Yolo416LayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool Yolo416LayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void Yolo416LayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void Yolo416LayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void Yolo416LayerPlugin::detachFromContext() {}

    const char* Yolo416LayerPlugin::getPluginType() const
    {
        return "Yolo416Layer_TRT";
    }

    const char* Yolo416LayerPlugin::getPluginVersion() const
    {
        return "1";
    }

    void Yolo416LayerPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* Yolo416LayerPlugin::clone() const
    {
        Yolo416LayerPlugin *p = new Yolo416LayerPlugin();
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data){ return 1./(1. + exp(-data)); };

    __global__ void CalYolo416Detection(const float *input, float *output,int noElements,
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes,int outputElem) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid*bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < 3; ++k) {
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH) continue;

            float *res_count = output + bnIdx*outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= MAX_OUTPUT_BBOX_COUNT) return;
            char* data = (char * )res_count + sizeof(float) + count*sizeof(Detection);
            Detection* det =  (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            det->bbox[0] = (col + Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * INPUT_W / yoloWidth;
            det->bbox[1] = (row + Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * INPUT_H / yoloHeight;
            det->bbox[2] = exp(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]) * anchors[2*k];
            det->bbox[3] = exp(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]) * anchors[2*k + 1];
            det->det_confidence = box_prob;
            det->class_id = class_id;
            det->class_confidence = max_cls_prob;
        }
    }

    void Yolo416LayerPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {

        int outputElem = 1 + MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        for(int idx = 0 ; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx*outputElem, 0, sizeof(float)));
        }
        int numElem = 0;
        for (unsigned int i = 0;i< mYoloKernel.size();++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height*batchSize;
            if (numElem < mThreadCount)
                mThreadCount = numElem;
            CalYolo416Detection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                (inputs[i],output, numElem, yolo.width, yolo.height, (float *)mAnchor[i], mClassCount ,outputElem);
        }

    }


    int Yolo416LayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);

        return 0;
    }

    PluginFieldCollection Yolo416PluginCreator::mFC{};
    std::vector<PluginField> Yolo416PluginCreator::mPluginAttributes;

    Yolo416PluginCreator::Yolo416PluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* Yolo416PluginCreator::getPluginName() const
    {
            return "Yolo416Layer_TRT";
    }

    const char* Yolo416PluginCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* Yolo416PluginCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* Yolo416PluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        Yolo416LayerPlugin* obj = new Yolo416LayerPlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* Yolo416PluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        Yolo416LayerPlugin* obj = new Yolo416LayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}